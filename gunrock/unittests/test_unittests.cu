// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_unitests.cu
 *
 * @brief Main test driver for all googletests.
 * @source
 * https://github.com/google/googletest/blob/master/googletest/docs/Primer.md
 */

#include <stdio.h>
#include <gunrock/gunrock.h>

#include <gunrock/app/hello/hello_app.cu>
#include <gunrock/app/test_base.cuh>

#include <gtest/gtest.h>

/**
 * @brief: Gunrock: Google tests -- list of tests
 * found in this directory, testing core functionality
 * of gunrock: primitives, operators, device intrinsics,
 * etc.
 *
 */

// bug:: malloc_consolidate(): invalid chunk size
//#include "test_lib_pr.h"

// Tests Subgraph Matching
#include "test_lib_sm.h"

// Tests the RepeatFor Operator
#include "test_repeatfor.h"

// Tests Segmented Reduction (device)
#include "test_segreduce.h"

// Tests Binary Search
#include "test_binarysearch.h"

using namespace gunrock;

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;

  GUARD_CU(parameters.Use<bool>("googletest", util::OPTIONAL_PARAMETER, true,
                                "Example parameter for googletest", __FILE__,
                                __LINE__));

  return retval;
}

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val) {
    // CLI parameters
    bool quick = parameters.Get<bool>("quick");
    bool quiet = parameters.Get<bool>("quiet");

    hipError_t retval = hipSuccess;
    return retval;
  }
};

int main(int argc, char **argv) {
  hipError_t retval = hipSuccess;

  util::Parameters parameters("test unittests");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(app::UseParameters_test(parameters));
  GUARD_CU(UseParameters(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }

  // Run all tests using the google tests
  // framework.
  ::testing::InitGoogleTest(&argc, argv);
  RUN_ALL_TESTS();

  return app::Switch_Types<app::VERTEXT_U32B | app::VERTEXT_U64B |
                           app::SIZET_U32B |   // app::SIZET_U64B |
                           app::VALUET_F32B |  // app::VALUET_F64B |
                           app::DIRECTED | app::UNDIRECTED>(parameters,
                                                            main_struct());
}
