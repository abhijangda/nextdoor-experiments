// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_knn.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <gunrock/app/knn/knn_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

namespace APP_NAMESPACE = app::knn;

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters& parameters, VertexT v, SizeT s, ValueT val) {
    // CLI parameters
    bool quick = parameters.Get<bool>("quick");
    bool quiet = parameters.Get<bool>("quiet");

    // Get number of nearest neighbors, default k = 10
    SizeT k = parameters.Get<int>("k");
    // Get x reference point, default point_id = 0
    VertexT point_x = parameters.Get<VertexT>("x");
    // Get y reference point, default point_id = 0
    VertexT point_y = parameters.Get<VertexT>("y");

    util::PrintMsg("Reference point is (" + std::to_string(point_x) + ", " +
                       std::to_string(point_y) + "), k = " + std::to_string(k) +
                       "\n",
                   !quiet);

    typedef typename app::TestGraph<VertexT, SizeT, ValueT, graph::HAS_CSR>
        GraphT;

    hipError_t retval = hipSuccess;
    util::CpuTimer cpu_timer;
    GraphT graph;

    cpu_timer.Start();
    GUARD_CU(graphio::LoadGraph(parameters, graph));
    cpu_timer.Stop();
    parameters.Set("load-time", cpu_timer.ElapsedMillis());

    // Reference result on CPU
    SizeT* ref_knns = NULL;
    SizeT* h_knns = (SizeT*)malloc(sizeof(SizeT) * graph.nodes * k);

    if (!quick) {
      // Init datastructures for reference result on GPU
      ref_knns = (SizeT*)malloc(sizeof(SizeT) * graph.nodes * k);

      // If not in `quick` mode, compute CPU reference implementation
      util::PrintMsg("__________________________", !quiet);
      util::PrintMsg("______ CPU Reference _____", !quiet);

      float elapsed = app::knn::CPU_Reference(graph.csr(), k, point_x, point_y,
                                              ref_knns, quiet);

      util::PrintMsg(
          "--------------------------\n Elapsed: " + std::to_string(elapsed),
          !quiet);
      util::PrintMsg("__________________________", !quiet);
      parameters.Set("cpu-elapsed", elapsed);
    }

    std::vector<std::string> switches{"advance-mode"};

    GUARD_CU(app::Switch_Parameters(
        parameters, graph, switches,
        [k, h_knns, ref_knns](util::Parameters& parameters, GraphT& graph) {
          return app::knn::RunTests(parameters, graph, k, h_knns, ref_knns,
                                    util::DEVICE);
        }));

    if (!quick) {
      delete[] ref_knns;
    }

    return retval;
  }
};

int main(int argc, char** argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test knn");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(app::knn::UseParameters(parameters));
  GUARD_CU(app::UseParameters_test(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  return app::Switch_Types<app::VERTEXT_U32B | app::VERTEXT_U64B |
                           app::SIZET_U32B | app::SIZET_U64B |
                           app::VALUET_F32B | app::DIRECTED | app::UNDIRECTED>(
      parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
