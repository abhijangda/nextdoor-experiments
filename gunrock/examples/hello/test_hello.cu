// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_hello.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <gunrock/app/hello/hello_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

namespace APP_NAMESPACE = app::hello;

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val) {
    // CLI parameters
    bool quick = parameters.Get<bool>("quick");
    bool quiet = parameters.Get<bool>("quiet");

    typedef typename app::TestGraph<VertexT, SizeT, ValueT, graph::HAS_CSR>
        GraphT;

    hipError_t retval = hipSuccess;
    util::CpuTimer cpu_timer;
    GraphT graph;

    cpu_timer.Start();
    GUARD_CU(graphio::LoadGraph(parameters, graph));
    cpu_timer.Stop();
    parameters.Set("load-time", cpu_timer.ElapsedMillis());

    // <TODO> get srcs if needed, e.g.:
    // GUARD_CU(app::Set_Srcs (parameters, graph));
    // std::vector<VertexT> srcs
    //    = parameters.Get<std::vector<VertexT> >("srcs");
    // int num_srcs = srcs.size();
    // </TODO>

    // <TODO> declare datastructures for reference result on GPU
    VertexT *ref_degrees;
    // </TODO>

    if (false && !quick) {
      // <TODO> init datastructures for reference result on GPU
      ref_degrees = new VertexT[graph.nodes];
      // </TODO>

      // If not in `quick` mode, compute CPU reference implementation
      util::PrintMsg("__________________________", !quiet);

      float elapsed =
          app::hello::CPU_Reference(graph.csr(), nullptr, quiet);

      util::PrintMsg(
          "--------------------------\n Elapsed: " + std::to_string(elapsed),
          !quiet);
    }

    // <TODO> add other switching parameters, if needed
    std::vector<std::string> switches{"advance-mode"};
    // </TODO>

    GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
                                    [
                                        // </TODO> pass necessary data to lambda
                                        ref_degrees
                                        // </TODO>
    ](util::Parameters &parameters, GraphT &graph) {
                                      // <TODO> pass necessary data to
                                      // app::Template::RunTests
                                      return app::hello::RunTests(
                                          parameters, graph, ref_degrees,
                                          util::DEVICE);
                                      // </TODO>
                                    }));

    if (!quick) {
      // <TODO> deallocate host references
      delete[] ref_degrees;
      ref_degrees = NULL;
      // </TODO>
    }
    return retval;
  }
};

int main(int argc, char **argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test hello");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(app::hello::UseParameters(parameters));
  GUARD_CU(app::UseParameters_test(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  // TODO: change available graph types, according to requirements
  return app::Switch_Types<app::VERTEXT_U32B | app::VERTEXT_U64B |
                           app::SIZET_U32B | app::SIZET_U64B |
                           app::VALUET_F32B | app::DIRECTED | app::UNDIRECTED>(
      parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
