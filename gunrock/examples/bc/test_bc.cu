// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bc.cu
 *
 * @brief Simple test driver program for Gunrock GC.
 */

#include <iostream>
#include <gunrock/app/bc/bc_app.cu>
#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val) {
    typedef typename app::TestGraph<VertexT, SizeT, ValueT, graph::HAS_CSR>
        GraphT;

    hipError_t retval = hipSuccess;
    util::CpuTimer cpu_timer;
    GraphT graph;

    cpu_timer.Start();
    GUARD_CU(graphio::LoadGraph(parameters, graph));
    cpu_timer.Stop();
    parameters.Set("load-time", cpu_timer.ElapsedMillis());

    // Enable is set sources
    GUARD_CU(app::Set_Srcs(parameters, graph));
    int num_srcs = 0;

    ValueT **reference_bc_values = NULL;
    ValueT **reference_sigmas = NULL;
    VertexT **reference_source_path = NULL;

    bool quick = parameters.Get<bool>("quick");
    bool quiet = parameters.Get<bool>("quiet");

    if (!quick) {
      // std::string validation = parameters.Get<std::string>("validation");
      util::PrintMsg("Computing reference value ...", !quiet);
      std::vector<VertexT> srcs = parameters.Get<std::vector<VertexT> >("srcs");
      num_srcs = srcs.size();

      SizeT nodes = graph.nodes;

      reference_bc_values = new ValueT *[num_srcs];
      reference_sigmas = new ValueT *[num_srcs];
      reference_source_path = new VertexT *[num_srcs];

      for (int i = 0; i < num_srcs; i++) {
        VertexT src = srcs[i];
        util::PrintMsg("__________________________", !quiet);

        reference_bc_values[i] = new ValueT[nodes];
        reference_sigmas[i] = new ValueT[nodes];
        reference_source_path[i] = new VertexT[nodes];

        float elapsed = app::bc::CPU_Reference(
            graph, reference_bc_values[i], reference_sigmas[i],
            reference_source_path[i], src, quiet);
        util::PrintMsg("--------------------------\nRun " + std::to_string(i) +
                           " elapsed: " + std::to_string(elapsed) +
                           " ms, src = " + std::to_string(src),
                       !quiet);
      }
    }

    std::vector<std::string> switches{"advance-mode"};
    GUARD_CU(app::Switch_Parameters(
        parameters, graph, switches,
        [reference_bc_values, reference_sigmas, reference_source_path](
            util::Parameters &parameters, GraphT &graph) {
          return app::bc::RunTests(parameters, graph, reference_bc_values,
                                   reference_sigmas, reference_source_path);
        }));

    // Cleanup
    if (!quick) {
      for (int i = 0; i < num_srcs; i++) {
        delete[] reference_bc_values[i];
        reference_bc_values[i] = NULL;
        delete[] reference_sigmas[i];
        reference_sigmas[i] = NULL;
        delete[] reference_source_path[i];
        reference_source_path[i] = NULL;
      }
      delete[] reference_bc_values;
      reference_bc_values = NULL;
      delete[] reference_sigmas;
      reference_sigmas = NULL;
      delete[] reference_source_path;
      reference_source_path = NULL;
    }
    return retval;
  }
};

int main(int argc, char **argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test bc");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(app::bc::UseParameters(parameters));
  GUARD_CU(app::UseParameters_test(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  return app::Switch_Types<app::VERTEXT_U32B |  // app::VERTEXT_U64B |
                           app::SIZET_U32B |    // app::SIZET_U64B |
                           app::VALUET_F32B | app::DIRECTED | app::UNDIRECTED>(
      parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
