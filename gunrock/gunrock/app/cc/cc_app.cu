// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file cc_app.cu
 *
 * @brief connected component (CC) application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// connected component includes
#include <gunrock/app/cc/cc_enactor.cuh>
#include <gunrock/app/cc/cc_problem.cuh>
#include <gunrock/app/cc/cc_functor.cuh>

#include <unistd.h>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::cc;

/**
 * @brief CC_Parameter structure
 */
struct CC_Parameter : gunrock::app::TestParameter_Base {
 public:
  CC_Parameter() {}
  ~CC_Parameter() {}
};

template <typename VertexId, typename SizeT, typename Value>
// bool INSTRUMENT,
// bool DEBUG,
// bool SIZE_CHECK >
void runCC(GRGraph *output, CC_Parameter *parameter);

/**
 * @brief Run test
 *
 * @tparam VertexId   Vertex identifier type
 * @tparam Value      Attribute type
 * @tparam SizeT      Graph size type
 * @tparam INSTRUMENT Keep kernels statics
 * @tparam DEBUG      Keep debug statics
 * @tparam SIZE_CHECK Enable size check
 *
 * @param[out] output    Pointer to output graph structure of the problem
 * @param[in]  parameter primitive-specific test parameters
 */
template <typename VertexId, typename SizeT, typename Value>
// bool INSTRUMENT,
// bool DEBUG,
// bool SIZE_CHECK >
void runCC(GRGraph *output, CC_Parameter *parameter) {
  typedef CCProblem<VertexId, SizeT,
                    Value> Problem;  // use double buffer

  typedef CCEnactor<Problem>
      // INSTRUMENT,
      // DEBUG,
      // SIZE_CHECK >
      Enactor;

  Csr<VertexId, SizeT, Value> *graph =
      (Csr<VertexId, SizeT, Value> *)parameter->graph;
  bool quiet = parameter->g_quiet;
  int max_grid_size = parameter->max_grid_size;
  int num_gpus = parameter->num_gpus;
  double max_queue_sizing = parameter->max_queue_sizing;
  double max_in_sizing = parameter->max_in_sizing;
  ContextPtr *context = (ContextPtr *)parameter->context;
  std::string partition_method = parameter->partition_method;
  int *gpu_idx = parameter->gpu_idx;
  hipStream_t *streams = parameter->streams;
  float partition_factor = parameter->partition_factor;
  int partition_seed = parameter->partition_seed;
  bool g_stream_from_host = parameter->g_stream_from_host;
  bool instrument = parameter->instrumented;
  bool debug = parameter->debug;
  bool size_check = parameter->size_check;
  std::string traversal_mode = parameter->traversal_mode;
  size_t *org_size = new size_t[num_gpus];
  // Allocate host-side label array
  VertexId *h_component_ids = new VertexId[graph->nodes];

  for (int gpu = 0; gpu < num_gpus; gpu++) {
    size_t dummy;
    hipSetDevice(gpu_idx[gpu]);
    hipMemGetInfo(&(org_size[gpu]), &dummy);
  }

  Problem *problem = new Problem;  // Allocate problem on GPU
  util::GRError(
      problem->Init(g_stream_from_host, graph, NULL, num_gpus, gpu_idx,
                    partition_method, streams, max_queue_sizing, max_in_sizing,
                    partition_factor, partition_seed),
      "CC Problem Initialization Failed", __FILE__, __LINE__);

  Enactor *enactor = new Enactor(num_gpus, gpu_idx, instrument, debug,
                                 size_check);  // CC enactor map

  util::GRError(enactor->Init(context, problem, traversal_mode, max_grid_size),
                "CC Enactor Init failed", __FILE__, __LINE__);

  // Perform CC
  CpuTimer cpu_timer;

  util::GRError(problem->Reset(enactor->GetFrontierType(), max_queue_sizing),
                "CC Problem Data Reset Failed", __FILE__, __LINE__);
  util::GRError(enactor->Reset(), "CC Enactor Reset failed", __FILE__,
                __LINE__);

  // printf("frontier type:%d\n", enactor->GetFrontierType());
  usleep(1000);
  cpu_timer.Start();
  util::GRError(enactor->Enact(traversal_mode), "CC Problem Enact Failed",
                __FILE__, __LINE__);
  cpu_timer.Stop();

  float elapsed = cpu_timer.ElapsedMillis();

  // Copy out results
  util::GRError(problem->Extract(h_component_ids),
                "CC Problem Data Extraction Failed", __FILE__, __LINE__);

  unsigned int num_components = problem->num_components;
  output->aggregation = (unsigned int *)&num_components;
  output->node_value1 = (VertexId *)&h_component_ids[0];

  if (!quiet) {
    printf(" GPU Connected Component finished in %lf msec.\n", elapsed);
  }

  // Clean up
  if (org_size) {
    delete[] org_size;
    org_size = NULL;
  }
  if (problem) {
    delete problem;
    problem = NULL;
  }
  if (enactor) {
    delete enactor;
    enactor = NULL;
  }
}

/**
 * @brief Dispatch function to handle configurations
 *
 * @param[out] grapho  Pointer to output graph structure of the problem
 * @param[in]  graphi  Pointer to input graph we need to process on
 * @param[in]  config  Primitive-specific configurations
 * @param[in]  data_t  Data type configurations
 * @param[in]  context ModernGPU context
 * @param[in]  streams CUDA stream
 */
void dispatch_cc(GRGraph *grapho, const GRGraph *graphi, const GRSetup *config,
                 const GRTypes data_t, ContextPtr *context,
                 hipStream_t *streams) {
  CC_Parameter *parameter = new CC_Parameter;
  parameter->context = context;
  parameter->streams = streams;
  parameter->g_quiet = config->quiet;
  parameter->num_gpus = config->num_devices;
  parameter->gpu_idx = config->device_list;

  switch (data_t.VTXID_TYPE) {
    case VTXID_INT: {
      switch (data_t.SIZET_TYPE) {
        case SIZET_INT: {
          switch (data_t.VALUE_TYPE) {
            case VALUE_INT:  // template type = <int, int, int>
            {
              // build input CSR format graph
              Csr<int, int, int> csr(false);
              csr.nodes = graphi->num_nodes;
              csr.edges = graphi->num_edges;
              csr.row_offsets = (int *)graphi->row_offsets;
              csr.column_indices = (int *)graphi->col_indices;
              parameter->graph = &csr;

              runCC<int, int, int>(grapho, parameter);

              // reset for free memory
              csr.row_offsets = NULL;
              csr.column_indices = NULL;
              break;
            }
            case VALUE_UINT:  // template type = <int, uint, int>
            {
              printf("Not Yet Support This DataType Combination.\n");
              break;
            }
            case VALUE_FLOAT:  // template type = <int, float, int>
            {
              printf("Not Yet Support This DataType Combination.\n");
              break;
            }
          }
          break;
        }
      }
      break;
    }
  }
}

/*
 * @brief Entry of gunrock_cc function
 *
 * @param[out] grapho Pointer to output graph structure of the problem
 * @param[in]  graphi Pointer to input graph we need to process on
 * @param[in]  config Gunrock primitive specific configurations
 * @param[in]  data_t Gunrock data type structure
 */
void gunrock_cc(GRGraph *grapho, const GRGraph *graphi, const GRSetup *config,
                const GRTypes data_t) {
  // GPU-related configurations
  int num_gpus = 0;
  int *gpu_idx = NULL;
  ContextPtr *context = NULL;
  hipStream_t *streams = NULL;

  num_gpus = config->num_devices;
  gpu_idx = new int[num_gpus];
  for (int i = 0; i < num_gpus; ++i) {
    gpu_idx[i] = config->device_list[i];
  }

  // Create streams and MordernGPU context for each GPU
  streams = new hipStream_t[num_gpus * num_gpus * 2];
  context = new ContextPtr[num_gpus * num_gpus];
  if (!config->quiet) {
    printf(" using %d GPUs:", num_gpus);
  }
  for (int gpu = 0; gpu < num_gpus; ++gpu) {
    if (!config->quiet) {
      printf(" %d ", gpu_idx[gpu]);
    }
    util::SetDevice(gpu_idx[gpu]);
    for (int i = 0; i < num_gpus * 2; ++i) {
      int _i = gpu * num_gpus * 2 + i;
      util::GRError(hipStreamCreate(&streams[_i]), "hipStreamCreate fialed.",
                    __FILE__, __LINE__);
      if (i < num_gpus) {
        context[gpu * num_gpus + i] =
            mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu], streams[_i]);
      }
    }
  }
  if (!config->quiet) {
    printf("\n");
  }

  dispatch_cc(grapho, graphi, config, data_t, context, streams);
}

/*
 * @brief Simple interface take in CSR arrays as input
 *
 * @param[out] components  Return component ID for each node
 * @param[out] num_comps   Return number of components calculated
 * @param[in]  num_nodes   Number of nodes of the input graph
 * @param[in]  num_edges   Number of edges of the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 */
int cc(int *component, const int num_nodes, const int num_edges,
       const int *row_offsets, const int *col_indices) {
  struct GRTypes data_t;          // primitive-specific data types
  data_t.VTXID_TYPE = VTXID_INT;  // integer vertex identifier
  data_t.SIZET_TYPE = SIZET_INT;  // integer graph size type
  data_t.VALUE_TYPE = VALUE_INT;  // integer attributes type

  struct GRSetup *config = InitSetup(1, NULL);  // primitive-specific configures

  struct GRGraph *grapho = (struct GRGraph *)malloc(sizeof(struct GRGraph));
  struct GRGraph *graphi = (struct GRGraph *)malloc(sizeof(struct GRGraph));

  graphi->num_nodes = num_nodes;                  // setting graph nodes
  graphi->num_edges = num_edges;                  // setting graph edges
  graphi->row_offsets = (void *)&row_offsets[0];  // setting row_offsets
  graphi->col_indices = (void *)&col_indices[0];  // setting col_indices

  gunrock_cc(grapho, graphi, config, data_t);
  int *num_components = (int *)grapho->aggregation;
  memcpy(component, (int *)grapho->node_value1, num_nodes * sizeof(int));

  if (graphi) free(graphi);
  if (grapho) free(grapho);

  return *num_components;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
