// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file mf_app.cu
 *
 * @brief maxflow (mf) application
 */

#include <gunrock/gunrock.h>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>
#include <gunrock/util/type_limits.cuh>

// Graph definations
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>
#include <gunrock/graphio/graphio.cuh>

// MF includes
#include <gunrock/app/mf/mf_enactor.cuh>
#include <gunrock/app/mf/mf_test.cuh>

//#define debug_aml(a...) {printf("%s:%d ", __FILE__, __LINE__); printf(a);\
    printf("\n");}
#define debug_aml(a...)

namespace gunrock {
namespace app {
namespace mf {

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));

  GUARD_CU(parameters.Use<uint64_t>(
      "source", util::REQUIRED_ARGUMENT | util::SINGLE_VALUE,
      util::PreDefinedValues<uint64_t>::InvalidValue,
      "<Vertex-ID|random|largestdegree> The source vertex\n"
      "\tIf random, randomly select non-zero degree vertex;\n"
      "\tIf largestdegree, select vertex with largest degree",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<uint64_t>(
      "sink", util::REQUIRED_ARGUMENT | util::SINGLE_VALUE,
      util::PreDefinedValues<uint64_t>::InvalidValue,
      "<Vertex-ID|random|largestdegree> The source vertex\n"
      "\tIf random, randomly select non-zero degree vertex;\n"
      "\tIf largestdegree, select vertex with largest degree",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "num-repeats",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      util::PreDefinedValues<int>::InvalidValue,
      "Number of repeats for ReapetFor operator\n"
      "\tDefault num-repeats is linear from number of vertices",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "seed",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      util::PreDefinedValues<int>::InvalidValue,
      "seed to generate random sources or sink", __FILE__, __LINE__));
  return retval;
}

/**
 * @brief Run mf tests
 * @tparam     GraphT	  Type of the graph
 * @tparam     ValueT	  Type of the capacity on edges
 * @tparam     VertexT	  Type of vertex
 * @param[in]  parameters Excution parameters
 * @param[in]  graph	  Input graph
 * @param[in]  ref_flow	  Reference flow on edges
 * @param[in]  target	  Whether to perform the mf
 * \return hipError_t error message(s), if any
 */
template <typename GraphT, typename ValueT, typename VertexT>
hipError_t RunTests(util::Parameters &parameters, GraphT &graph,
                     VertexT *h_reverse, ValueT *ref_flow, ValueT ref_max_flow,
                     util::Location target = util::DEVICE) {
  debug_aml("RunTests starts");
  hipError_t retval = hipSuccess;

  typedef Problem<GraphT> ProblemT;
  typedef Enactor<ProblemT> EnactorT;

  util::CpuTimer total_timer;
  total_timer.Start();
  util::CpuTimer cpu_timer;
  cpu_timer.Start();

  // parse configurations from parameters
  bool quiet_mode = parameters.Get<bool>("quiet");
  int num_runs = parameters.Get<int>("num-runs");
  std::string validation = parameters.Get<std::string>("validation");
  VertexT source = parameters.Get<VertexT>("source");
  VertexT sink = parameters.Get<VertexT>("sink");
  int num_repeats = parameters.Get<int>("num-repeats");
  debug_aml("source %d, sink %d, quite_mode %d, num-runs %d", source, sink,
            quiet_mode, num_runs);

  util::Info info("MF", parameters, graph);  // initialize Info structure

  // Allocate host-side array (for both reference and GPU-computed results)
  // ... for function Extract

  ValueT *h_flow = new ValueT[graph.edges];
  int *min_cut = new int[graph.nodes];
  // for (auto u = 0; u < graph.nodes; ++u) min_cut[u] = 0;
  memset(min_cut, 0, graph.nodes * sizeof(min_cut[0]));

  bool *vertex_reachabilities = new bool[graph.nodes];

  ValueT *h_residuals = new ValueT[graph.edges];

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  GUARD_CU(problem.Init(graph, target));
  GUARD_CU(enactor.Init(problem, target));

  cpu_timer.Stop();
  parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());

  // perform the MF algorithm
  for (int run_num = 0; run_num < num_runs; ++run_num) {
    GUARD_CU(problem.Reset(graph, h_reverse, target));
    GUARD_CU(enactor.Reset(source, target));

    util::PrintMsg("______GPU PushRelabel algorithm____", !quiet_mode);

    cpu_timer.Start();
    GUARD_CU(enactor.Enact());
    cpu_timer.Stop();
    info.CollectSingleRun(cpu_timer.ElapsedMillis());

    //    fprintf(stderr, "-----------------------------------\nRun %d, elapsed: %lf ms, #iterations = %d\n", \
		    run_num, cpu_timer.ElapsedMillis(), enactor.enactor_slices[0].enactor_stats.iteration);

    fprintf(stderr, "GPU Elapsed: %lf ms, ", cpu_timer.ElapsedMillis());
    util::PrintMsg(
        "-----------------------------------\nRun " + std::to_string(run_num) +
            ", elapsed: " + std::to_string(cpu_timer.ElapsedMillis()) +
            " ms, #iterations = " +
            std::to_string(enactor.enactor_slices[0].enactor_stats.iteration),
        !quiet_mode);
    if (validation == "each") {
      GUARD_CU(problem.Extract(h_flow));
      GUARD_CU2(hipDeviceSynchronize(), "hipDeviceSynchronize failed.");
      app::mf::minCut(graph, source, h_flow, min_cut, vertex_reachabilities,
                      h_residuals);
      GUARD_CU2(hipDeviceSynchronize(), "hipDeviceSynchronize failed.");
      int num_errors = app::mf::Validate_Results(
          parameters, graph, source, sink, h_flow, h_reverse, min_cut,
          ref_max_flow, ref_flow, quiet_mode);
    }
  }

  // Copy out results
  cpu_timer.Start();
  if (validation == "last") {
    GUARD_CU(problem.Extract(h_flow));
    GUARD_CU2(hipDeviceSynchronize(), "hipDeviceSynchronize failed.");

    app::mf::minCut(graph, source, h_flow, min_cut, vertex_reachabilities,
                    h_residuals);
    GUARD_CU2(hipDeviceSynchronize(), "hipDeviceSynchronize failed.");

    int num_errors = app::mf::Validate_Results(
        parameters, graph, source, sink, h_flow, h_reverse, min_cut,
        ref_max_flow, ref_flow, quiet_mode);
  }

// Compute running statistics
// info.ComputeTraversalStats(enactor, h_flow);

// Display_Memory_Usage(problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
  // Display_Performance_Profiling(&enactor);
#endif

  // Clean up
  GUARD_CU(enactor.Release(target));
  GUARD_CU(problem.Release(target));

  delete[] h_flow;
  h_flow = NULL;
  delete[] min_cut;
  min_cut = NULL;

  cpu_timer.Stop();
  total_timer.Stop();

  info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());

  return retval;
}

}  // namespace mf
}  // namespace app
}  // namespace gunrock

/*
 * @brief Entry of gunrock_maxflow function
 * @tparam     GraphT     Type of the graph
 * @tparam     ValueT     Type of the capacity/flow/excess
 *
 * @param[in]  parameters Excution parameters
 * @param[in]  graph      Input graph
 * @param[out] flow	  Return flow on edges
 * @param[out] maxflow	  Return flow value
 * @param[out] min_cut	  Return partition into two sets of nodes
 * \return     double     Return accumulated elapsed times for all runs
 */
#if 0
template <typename GraphT, typename VertexT = typename GraphT::VertexT,
    typename ValueT = typename GraphT::ValueT>

double gunrock_mf(
    gunrock::util::Parameters &parameters,
    GraphT  &graph,
    VertexT *reverse,
    ValueT  *flow,
    int	    *min_cut,
    ValueT  &maxflow,
    bool   *vertex_reachabilities,
    ValueT *h_residuals)
{
    typedef gunrock::app::mf::Problem<GraphT>	ProblemT;
    typedef gunrock::app::mf::Enactor<ProblemT> EnactorT;

    gunrock::util::CpuTimer cpu_timer;
    gunrock::util::Location target = gunrock::util::DEVICE;

    double total_time = 0;
    if (parameters.UseDefault("quiet"))
        parameters.Set("quiet", true);

    // Allocate problem and enactor on GPU, and initialize them
    ProblemT problem(parameters);
    EnactorT enactor;
    problem.Init(graph,	  target);
    enactor.Init(problem, target);

    int num_runs = parameters.Get<int>("num-runs");
    int source = parameters.Get<VertexT>("source");
    int sink = parameters.Get<VertexT>("sink");

    for (int run_num = 0; run_num < num_runs; ++run_num)
    {
        problem.Reset(graph, reverse, target);
        enactor.Reset(source, target);

        cpu_timer.Start();
        enactor.Enact();
        cpu_timer.Stop();

        total_time += cpu_timer.ElapsedMillis();
        problem.Extract(flow);
	    gunrock::app::mf::minCut(graph, source, flow, min_cut, vertex_reachabilities, h_residuals);
    }

    enactor.Release(target);
    problem.Release(target);
    return total_time;
}
#endif

/*
 * @brief Simple interface  take in graph as CSR format
 * @param[in]  num_nodes    Number of veritces in the input graph
 * @param[in]  num_edges    Number of edges in the input graph
 * @param[in]  row_offsets  CSR-formatted graph input row offsets
 * @param[in]  col_indices  CSR-formatted graph input column indices
 * @param[in]  capacity	    CSR-formatted graph input edge weights
 * @param[in]  num_runs     Number of runs to perform mf
 * @param[in]  source	    Source to push flow towards the sink
 * @param[out] flow	    Return flow calculated on edges
 * @param[out] maxflow	    Return maxflow value
 * \return     double       Return accumulated elapsed times for all runs
 */
/*
template <
    typename VertexT  = uint32_t,
    typename SizeT    = uint32_t,
    typename ValueT   = double>
double mf(
        const int     num_runs,
        ValueT	      *flow,
        ValueT	      &maxflow,
        int	      *min_cut,
        int	      undirected = 0
        )
{
    typedef typename gunrock::app::TestGraph<VertexT, SizeT, ValueT,
        gunrock::graph::HAS_EDGE_VALUES | gunrock::graph::HAS_CSR>  GraphT;
    typedef typename GraphT::CsrT				    CsrT;

    // Setup parameters
    gunrock::util::Parameters parameters("mf");
    gunrock::graphio::UseParameters(parameters);
    gunrock::app::mf::UseParameters(parameters);
    gunrock::app::UseParameters_test(parameters);
    parameters.Parse_CommandLine(0, NULL);
    parameters.Set("num-runs", num_runs);

    bool quiet = parameters.Get<bool>("quiet");

    GraphT d_graph;
    if (not undirected){
        parameters.Set<int>("remove-duplicate-edges", false);
        debug_aml("Load directed graph");
        gunrock::graphio::LoadGraph(parameters, d_graph);
    }

    GraphT u_graph;
    parameters.Set<int>("undirected", 1);
    parameters.Set<int>("remove-duplicate-edges", true);
    debug_aml("Load undirected graph");
    gunrock::graphio::LoadGraph(parameters, u_graph);

    if (parameters.Get<VertexT>("source") ==
            gunrock::util::PreDefinedValues<VertexT>::InvalidValue){
        parameters.Set("source", 0);
    }
    if (parameters.Get<VertexT>("sink") ==
            gunrock::util::PreDefinedValues<VertexT>::InvalidValue){
        parameters.Set("sink", u_graph.nodes-1);
    }

    VertexT* reverse = (VertexT*)malloc(sizeof(VertexT) * u_graph.edges);

    // Initialize reverse array.
    for (auto u = 0; u < u_graph.nodes; ++u)
    {
        auto e_start = u_graph.CsrT::GetNeighborListOffset(u);
        auto num_neighbors = u_graph.CsrT::GetNeighborListLength(u);
        auto e_end = e_start + num_neighbors;
        for (auto e = e_start; e < e_end; ++e)
        {
            auto v = u_graph.CsrT::GetEdgeDest(e);
            auto f_start = u_graph.CsrT::GetNeighborListOffset(v);
            auto num_neighbors2 = u_graph.CsrT::GetNeighborListLength(v);
            auto f_end = f_start + num_neighbors2;
            for (auto f = f_start; f < f_end; ++f)
            {
                auto z = u_graph.CsrT::GetEdgeDest(f);
                if (z == u)
                {
                    reverse[e] = f;
                    reverse[f] = e;
                    break;
                }
            }
        }
    }

    if (not undirected){
        // Correct capacity values on reverse edges
        for (auto u = 0; u < u_graph.nodes; ++u)
        {
            auto e_start = u_graph.CsrT::GetNeighborListOffset(u);
            auto num_neighbors = u_graph.CsrT::GetNeighborListLength(u);
            auto e_end = e_start + num_neighbors;
            for (auto e = e_start; e < e_end; ++e)
            {
                u_graph.CsrT::edge_values[e] = (ValueT)0;
                auto v = u_graph.CsrT::GetEdgeDest(e);
                // Looking for edge u->v in directed graph
                auto f_start = d_graph.CsrT::GetNeighborListOffset(u);
                auto num_neighbors2 = d_graph.CsrT::GetNeighborListLength(u);
                auto f_end = f_start + num_neighbors2;
                for (auto f = f_start; f < f_end; ++f)
                {
                    auto z = d_graph.CsrT::GetEdgeDest(f);
                    if (z == v and d_graph.CsrT::edge_values[f] > 0)
                    {
                        u_graph.CsrT::edge_values[e]  =
                            d_graph.CsrT::edge_values[f];
                        break;
                    }
                }
            }
        }
    }

    gunrock::util::Location target = gunrock::util::HOST;

    // Run the MF
    double elapsed_time = gunrock_mf(parameters, u_graph, reverse, flow,
            min_cut, maxflow);

    // Cleanup
    u_graph.Release();
    d_graph.Release();

    return elapsed_time;
}*/

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
