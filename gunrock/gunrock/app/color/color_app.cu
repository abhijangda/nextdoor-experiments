#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file color_app.cu
 *
 * @brief Graph Coloring Gunrock Application
 */

#include <gunrock/gunrock.h>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph definitions
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>
#include <gunrock/graphio/graphio.cuh>

// Graph Coloring
#include <gunrock/app/color/color_enactor.cuh>
#include <gunrock/app/color/color_test.cuh>

// Others
#include <cstdio>

namespace gunrock {
namespace app {
namespace color {

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));
  GUARD_CU(UseParameters_test(parameters));

  GUARD_CU(parameters.Use<unsigned int>(
        "num-colors",
        util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::INTERNAL_PARAMETER,
        0, "number of output colors", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "loop-color", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, true,
      "Serially compare rand to all node neighbor, disable to use advance \
      neighbor reduce (default=false)",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "min-color", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, true,
      "Enable coloring with minimum independent set as well as \
      maximum(default=true)",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "test-run", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, true,
      "Perform test run to atomically generate max iteration (default=true)",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "user-iter",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      3, "Number of iterations color should run for (default=3).", __FILE__,
      __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "JPL", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, false,
      "Use JPL exact coloring method (true=use JPL).", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "no-conflict", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, 0,
      "Resolve color conflict, 0 to skip check, 1 to check at end of\
      every iteration with random,\
      2 to check at end of every iteration with degree(default = 0).",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "prohibit-size", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, 0,
      "Needed to allocate memory for hash function, if parameter is\
      positive,\
      hash coloring is used instead of random coloring (default = 0).",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "seed", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, time(NULL),
      "seed for random number generator", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<bool>(
      "LBCOLOR", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, false,
      "load balancing enabled for graph coloring (true=neighbor_reduce)",
      __FILE__, __LINE__));

  return retval;
}

/**
 * @brief Run color tests
 * @tparam     GraphT        Type of the graph
 * @tparam     ValueT        Type of the distances
 * @param[in]  parameters    Excution parameters
 * @param[in]  graph         Input graph
...
 * @param[in]  target        where to perform the app
 * \return hipError_t error message(s), if any
 */
template <typename GraphT>
hipError_t RunTests(util::Parameters &parameters, GraphT &graph,
                     bool color_balance, typename GraphT::VertexT *ref_colors,
                     util::Location target) {
  hipError_t retval = hipSuccess;

  typedef typename GraphT::VertexT VertexT;
  typedef typename GraphT::ValueT ValueT;
  typedef typename GraphT::SizeT SizeT;
  typedef Problem<GraphT> ProblemT;
  typedef Enactor<ProblemT> EnactorT;

  // CLI parameters
  bool quiet_mode = parameters.Get<bool>("quiet");
  int num_runs = parameters.Get<int>("num-runs");
  std::string validation = parameters.Get<std::string>("validation");
  util::Info info("color", parameters, graph);

  util::CpuTimer cpu_timer, total_timer;
  cpu_timer.Start();
  total_timer.Start();

  VertexT *h_colors = new VertexT[graph.nodes];

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  GUARD_CU(problem.Init(graph, target));
  GUARD_CU(enactor.Init(problem, target));

  cpu_timer.Stop();
  parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());
  int num_colors = 0;
  for (int run_num = 0; run_num < num_runs; ++run_num) {
    GUARD_CU(problem.Reset(target));
    GUARD_CU(enactor.Reset(target));

    util::PrintMsg("__________________________", !quiet_mode);

    cpu_timer.Start();
    GUARD_CU(enactor.Enact());
    cpu_timer.Stop();
    info.CollectSingleRun(cpu_timer.ElapsedMillis());

    util::PrintMsg(
        "--------------------------\nRun " + std::to_string(run_num) +
            " elapsed: " + std::to_string(cpu_timer.ElapsedMillis()) +
            ", #iterations = " +
            std::to_string(enactor.enactor_slices[0].enactor_stats.iteration),
        !quiet_mode);
    if (validation == "each") {
      GUARD_CU(problem.Extract(h_colors));
      SizeT num_errors = Validate_Results(parameters, graph, h_colors,
                                          ref_colors, false);
    }
  }

  cpu_timer.Start();

  GUARD_CU(problem.Extract(h_colors));
  if (validation == "last") {
    SizeT num_errors = Validate_Results(parameters, graph, h_colors, ref_colors,
                                        false);
  }

  // count number of colors
  std::unordered_set<int> set;
  for (SizeT v = 0; v < graph.nodes; v++) {
    int c = h_colors[v];
    if (set.find(c) == set.end()) {
      set.insert(c);
      num_colors++;
    }
  }

  util::PrintMsg("Number of colors: " + std::to_string(num_colors), !quiet_mode);

  parameters.Set("num-colors", num_colors);

  // compute running statistics
  info.ComputeTraversalStats(enactor, (VertexT *)NULL);
// Display_Memory_Usage(problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
  // Display_Performance_Profiling(&enactor);
#endif

  // Clean up
  GUARD_CU(enactor.Release(target));
  GUARD_CU(problem.Release(target));
  delete[] h_colors;
  h_colors = NULL;
  cpu_timer.Stop();
  total_timer.Stop();

  info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
  return retval;
}

}  // namespace color
}  // namespace app
}  // namespace gunrock

/*
 * @brief Entry of gunrock_color function
 * @tparam     GraphT     Type of the graph
 * @tparam     VertexT    Type of the colors
 * @param[in]  parameters Excution parameters
 * @param[in]  graph      Input graph
 * @param[out] colors     Return generated colors for each run
 * @param[out] num_colors Return number of colors generated for each run
 * \return     double     Return accumulated elapsed times for all runs
 */
template <typename GraphT, typename VertexT = typename GraphT::VertexT,
          typename SizeT = typename GraphT::SizeT>
double gunrock_color(gunrock::util::Parameters &parameters, GraphT &graph,
                     VertexT **colors, SizeT *num_colors) {
  typedef gunrock::app::color::Problem<GraphT> ProblemT;
  typedef gunrock::app::color::Enactor<ProblemT> EnactorT;
  gunrock::util::CpuTimer cpu_timer;
  gunrock::util::Location target = gunrock::util::DEVICE;
  double total_time = 0;
  if (parameters.UseDefault("quiet")) parameters.Set("quiet", true);

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  problem.Init(graph, target);
  enactor.Init(problem, target);

  int num_runs = parameters.Get<int>("num-runs");
  for (int run_num = 0; run_num < num_runs; ++run_num) {
    problem.Reset(target);
    enactor.Reset(target);

    cpu_timer.Start();
    enactor.Enact();
    cpu_timer.Stop();

    total_time += cpu_timer.ElapsedMillis();
    problem.Extract(colors[run_num]);

    // count number of colors
    std::unordered_set<int> set;
    for (SizeT v = 0; v < graph.nodes; v++) {
      int c = colors[run_num][v];
      if (set.find(c) == set.end()) {
        set.insert(c);
        num_colors[run_num] += 1;
      }
    }
  }

  enactor.Release(target);
  problem.Release(target);
  return total_time;
}

/*
 * @brief Entry of gunrock_color function
 * @tparam     VertexT    Type of the colors
 * @tparam     SizeT      Type of the num_colors
 * @param[in]  parameters Excution parameters
 * @param[in]  graph      Input graph
 * @param[out] colors     Return generated colors for each run
 * @param[out] num_colors Return number of colors generated for each run
 * \return     double     Return accumulated elapsed times for all runs
 */
template <typename VertexT = int, typename SizeT = int,
          typename GValueT = unsigned int>
double color(const SizeT num_nodes, const SizeT num_edges,
             const SizeT *row_offsets, const VertexT *col_indices,
             const int num_runs, int **colors, int *num_colors,
             const GValueT edge_values = NULL) {
  typedef typename gunrock::app::TestGraph<VertexT, SizeT, GValueT,
                                           gunrock::graph::HAS_CSR>
      GraphT;
  typedef typename GraphT::CsrT CsrT;

  // Setup parameters
  gunrock::util::Parameters parameters("color");
  gunrock::graphio::UseParameters(parameters);
  gunrock::app::color::UseParameters(parameters);
  gunrock::app::UseParameters_test(parameters);
  parameters.Parse_CommandLine(0, NULL);
  parameters.Set("graph-type", "by-pass");
  parameters.Set("num-runs", num_runs);

  bool quiet = parameters.Get<bool>("quiet");
  GraphT graph;
  // Assign pointers into gunrock graph format
  graph.CsrT::Allocate(num_nodes, num_edges, gunrock::util::HOST);
  graph.CsrT::row_offsets.SetPointer((SizeT *)row_offsets, num_nodes + 1,
                                     gunrock::util::HOST);
  graph.CsrT::column_indices.SetPointer((VertexT *)col_indices, num_edges,
                                        gunrock::util::HOST);
  // graph.FromCsr(graph.csr(), true, quiet);
  gunrock::graphio::LoadGraph(parameters, graph);

  // Run the graph coloring
  double elapsed_time = gunrock_color(parameters, graph, colors, num_colors);

  // Cleanup
  graph.Release();

  return elapsed_time;
}

/*
 * @brief Entry of gunrock_color function
 * @tparam     VertexT    Type of the colors
 * @tparam     SizeT      Type of the num_colors
 * @param[in]  parameters Excution parameters
 * @param[in]  graph      Input graph
 * @param[out] colors     Return generated colors for each run
 * @param[out] num_colors Return number of colors generated for each run
 * \return     double     Return accumulated elapsed times for all runs
 */
double color(const int num_nodes, const int num_edges, const int *row_offsets,
             const int *col_indices, int *colors, int num_colors) {
  return color(num_nodes, num_edges, row_offsets, col_indices, 1 /* num_runs */,
               &colors, &num_colors);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
