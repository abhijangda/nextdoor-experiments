// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file hello_app.cu
 *
 * @brief Simple Gunrock Application
 */

#include <gunrock/gunrock.h>
#include <gunrock/util/test_utils.cuh>
#include <gunrock/graphio/graphio.cuh>
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>

#define S1 (10)
#define S2 (5)

// <TODO> change includes
#include <gunrock/app/graphsage-k-hop/hello_enactor.cuh>
#include <gunrock/app/graphsage-k-hop/hello_test.cuh>
// </TODO>

namespace gunrock {
namespace app {
// <TODO> change namespace
namespace hello {
// </TODO>

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));

  // <TODO> add app specific parameters, eg:
  // GUARD_CU(parameters.Use<std::string>(
  //    "src",
  //    util::REQUIRED_ARGUMENT | util::MULTI_VALUE | util::OPTIONAL_PARAMETER,
  //    "0",
  //    "<Vertex-ID|random|largestdegree> The source vertices\n"
  //    "\tIf random, randomly select non-zero degree vertices;\n"
  //    "\tIf largestdegree, select vertices with largest degrees",
  //    __FILE__, __LINE__));
  // </TODO>

  return retval;
}

/**
 * @brief Run hello tests
 * @tparam     GraphT        Type of the graph
 * @tparam     ValueT        Type of the distances
 * @param[in]  parameters    Excution parameters
 * @param[in]  graph         Input graph
...
 * @param[in]  target        where to perform the app
 * \return hipError_t error message(s), if any
 */
template <typename GraphT>
hipError_t RunTests(util::Parameters &parameters, GraphT &graph,
                     // <TODO> add problem specific reference results, e.g.:
                     typename GraphT::VertexT *ref_neighbors,
                     // </TODO>
                     util::Location target) {
  hipError_t retval = hipSuccess;

  typedef typename GraphT::VertexT VertexT;
  typedef typename GraphT::ValueT ValueT;
  typedef typename GraphT::SizeT SizeT;
  typedef typename GraphT::CsrT CsrT;
  typedef Problem<GraphT> ProblemT;
  typedef Enactor<ProblemT> EnactorT;

  // CLI parameters
  bool quiet_mode = parameters.Get<bool>("quiet");
  int num_runs = parameters.Get<int>("num-runs");
  std::string validation = parameters.Get<std::string>("validation");
  std::cout << __LINE__ << ": " << graph.nodes << std::endl;
  util::Info info("hello", parameters, graph);

  util::CpuTimer cpu_timer, total_timer;
  cpu_timer.Start();
  total_timer.Start();

  // <TODO> get problem specific inputs, e.g.:
  // std::vector<VertexT> srcs = parameters.Get<std::vector<VertexT>>("srcs");
  // printf("RunTests: %d srcs: src[0]=%d\n", srcs.size(), srcs[0]);
  // </TODO>

  // <TODO> allocate problem specific host data, e.g.:
  VertexT *h_neighbors = new VertexT[graph.edges];
  SizeT *h_positions = new SizeT[graph.nodes];
  SizeT *h_lengths = new SizeT[graph.nodes];
  SizeT *h_total_lengths = new SizeT[1];
  // </TODO>

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  GUARD_CU(problem.Init(graph, target));
  GUARD_CU(problem.InitNeighborsForHop (graph.nodes*S1, 0, target));
  GUARD_CU(enactor.Init(problem, target));

  cpu_timer.Stop();
  parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());
  graph.Display ();
  for (int run_num = 0; run_num < num_runs; ++run_num) {
    GUARD_CU(problem.Reset(
        // <TODO> problem specific data if necessary, eg:
        // src,
        // </TODO>
        target));
    GUARD_CU(enactor.Reset(
        // <TODO> problem specific data if necessary:
        // srcs[run_num % srcs.size()],
        // </TODO>
        target));

    util::PrintMsg("__________________________", !quiet_mode);

    cpu_timer.Start();
    for (int __hop = 0; __hop < N_HOPS; __hop++) {
      GUARD_CU(enactor.Enact(
          __hop
          ));
    }
    cpu_timer.Stop();
    info.CollectSingleRun(cpu_timer.ElapsedMillis());

    util::PrintMsg(
        "--------------------------\nRun " + std::to_string(run_num) +
            " elapsed: " + std::to_string(cpu_timer.ElapsedMillis()) +
            ", #iterations = " +
            std::to_string(enactor.enactor_slices[0].enactor_stats.iteration),
        !quiet_mode);

    if (false && validation == "each") {
      // GUARD_CU(problem.Extract(
      //     // <TODO> problem specific data
      //     h_neighbors,
      //     h_positions,
      //     h_lengths,
      //     h_total_lengths
      //     // </TODO>
      //     ));
      // // SizeT num_errors = Validate_Results(parameters, graph,
      //                                     // <TODO> problem specific data
      //                                     h_neighbors, ref_neighbors,
      //                                     // </TODO>
      //                                     false);
    }
  }

  cpu_timer.Start();

  //for (int hop = 0; hop < N_HOPS; hop++) 
  int hop = 1;
  {
    printf ("for hop %d\n", hop);
    SizeT* h_lengths = new SizeT[graph.nodes];

    GUARD_CU(problem.Extract_total_lengths(
      // <TODO> problem specific data
      hop,
      h_total_lengths
      // </TODO>
      ));
    //h_neighbors = new VertexT[h_total_lengths];

    // GUARD_CU(problem.Extract(
    //     // <TODO> problem specific data
    //     hop,
    //     h_neighbors,
    //     h_total_lengths,
    //     h_positions,
    //     h_lengths,
    //     &h_total_lengths
    //     // </TODO>
    //     ));
    
    int sum_e = 0;
    for (SizeT v = 0; v < graph.nodes; v++) {
      //printf ("v %d pos %d l %d\n", v, h_positions[v], h_lengths[v]);
      sum_e += h_lengths[v];
    }

    std::cout << "Sum of lengths " << sum_e << std::endl;
    // for (SizeT v = 0; v < graph.nodes - 1; v++) {
    //   //printf ("v %d l1 %d pos %d\n", v, h_lengths[v], h_positions[v]); //graph.CsrT::GetNeighborListOffset (v+1)-graph.CsrT::GetNeighborListOffset (v),);
    //   //assert (h_lengths[v] == graph.CsrT::GetNeighborListOffset (v+1)-graph.CsrT::GetNeighborListOffset (v));
    //   for (int e = 0; e < h_lengths[v]; e++) {
    //     int d_pos = h_positions[v] + e;
    //     int cpu_pos = graph.CsrT::GetNeighborListOffset (v) + e;
    //    // printf ("   %d %d\n", h_neighbors[d_pos], graph.CsrT::GetEdgeDest (cpu_pos));
    //   }
    // }
  }

  std::cout << "Time to exclude " << enactor.exclude_time << std::endl;

  if (false && validation == "last") {
    // SizeT num_errors = Validate_Results(parameters, graph,
    //                                     // <TODO> problem specific data
    //                                     h_neighbors, ref_neighbors,
    //                                     // </TODO>
    //                                     false);
  }

  // compute running statistics
  // <TODO> change NULL to problem specific per-vertex visited marker, e.g.
  // h_distances
  info.ComputeTraversalStats(enactor, (VertexT *)NULL);
// Display_Memory_Usage(problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
  // Display_Performance_Profiling(&enactor);
#endif
  // </TODO>

  // Clean up
  GUARD_CU(enactor.Release(target));
  GUARD_CU(problem.Release(target));
  // <TODO> Release problem specific data, e.g.:
  delete[] h_neighbors;
  h_neighbors = NULL;
  // </TODO>
  cpu_timer.Stop();
  total_timer.Stop();

  info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
  return retval;
}

}  // namespace hello
}  // namespace app
}  // namespace gunrock

// ===========================================================================================
// ========================= CODE BELOW THIS LINE NOT NEEDED FOR TESTS
// =======================
// ===========================================================================================

// /*
// * @brief Entry of gunrock_template function
// * @tparam     GraphT     Type of the graph
// * @tparam     ValueT     Type of the distances
// * @param[in]  parameters Excution parameters
// * @param[in]  graph      Input graph
// * @param[out] distances  Return shortest distance to source per vertex
// * @param[out] preds      Return predecessors of each vertex
// * \return     double     Return accumulated elapsed times for all runs
// */
// template <typename GraphT, typename ValueT = typename GraphT::ValueT>
// double gunrock_Template(
//     gunrock::util::Parameters &parameters,
//     GraphT &graph
//     // TODO: add problem specific outputs, e.g.:
//     //ValueT **distances
//     )
// {
//     typedef typename GraphT::VertexT VertexT;
//     typedef gunrock::app::Template::Problem<GraphT  > ProblemT;
//     typedef gunrock::app::Template::Enactor<ProblemT> EnactorT;
//     gunrock::util::CpuTimer cpu_timer;
//     gunrock::util::Location target = gunrock::util::DEVICE;
//     double total_time = 0;
//     if (parameters.UseDefault("quiet"))
//         parameters.Set("quiet", true);

//     // Allocate problem and enactor on GPU, and initialize them
//     ProblemT problem(parameters);
//     EnactorT enactor;
//     problem.Init(graph  , target);
//     enactor.Init(problem, target);

//     int num_runs = parameters.Get<int>("num-runs");
//     // TODO: get problem specific inputs, e.g.:
//     // std::vector<VertexT> srcs =
//     parameters.Get<std::vector<VertexT>>("srcs");
//     // int num_srcs = srcs.size();
//     for (int run_num = 0; run_num < num_runs; ++run_num)
//     {
//         // TODO: problem specific inputs, e.g.:
//         // int src_num = run_num % num_srcs;
//         // VertexT src = srcs[src_num];
//         problem.Reset(/*src,*/ target);
//         enactor.Reset(/*src,*/ target);

//         cpu_timer.Start();
//         enactor.Enact(/*src*/);
//         cpu_timer.Stop();

//         total_time += cpu_timer.ElapsedMillis();
//         // TODO: extract problem specific data, e.g.:
//         problem.Extract(/*distances[src_num]*/);
//     }

//     enactor.Release(target);
//     problem.Release(target);
//     // TODO: problem specific clean ups, e.g.:
//     // srcs.clear();
//     return total_time;
// }

//  * @brief Simple interface take in graph as CSR format
//  * @param[in]  num_nodes   Number of veritces in the input graph
//  * @param[in]  num_edges   Number of edges in the input graph
//  * @param[in]  row_offsets CSR-formatted graph input row offsets
//  * @param[in]  col_indices CSR-formatted graph input column indices
//  * @param[in]  edge_values CSR-formatted graph input edge weights
//  * @param[in]  num_runs    Number of runs to perform SSSP
//  * @param[in]  sources     Sources to begin traverse, one for each run
//  * @param[in]  mark_preds  Whether to output predecessor info
//  * @param[out] distances   Return shortest distance to source per vertex
//  * @param[out] preds       Return predecessors of each vertex
//  * \return     double      Return accumulated elapsed times for all runs

// template <
//     typename VertexT = int,
//     typename SizeT   = int,
//     typename GValueT = unsigned int,
//     typename TValueT = GValueT>
// float Template(
//     const SizeT        num_nodes,
//     const SizeT        num_edges,
//     const SizeT       *row_offsets,
//     const VertexT     *col_indices,
//     const GValueT     *edge_values,
//     const int          num_runs
//     // TODO: add problem specific inputs and outputs, e.g.:
//     //      VertexT     *sources,
//     //      SSSPValueT **distances
//     )
// {
//     // TODO: change to other graph representation, if not using CSR
//     typedef typename gunrock::app::TestGraph<VertexT, SizeT, GValueT,
//         gunrock::graph::HAS_EDGE_VALUES | gunrock::graph::HAS_CSR>
//         GraphT;
//     typedef typename GraphT::CsrT CsrT;

//     // Setup parameters
//     gunrock::util::Parameters parameters("Template");
//     gunrock::graphio::UseParameters(parameters);
//     gunrock::app::Template::UseParameters(parameters);
//     gunrock::app::UseParameters_test(parameters);
//     parameters.Parse_CommandLine(0, NULL);
//     parameters.Set("graph-type", "by-pass");
//     parameters.Set("num-runs", num_runs);
//     // TODO: problem specific inputs, e.g.:
//     // std::vector<VertexT> srcs;
//     // for (int i = 0; i < num_runs; i ++)
//     //     srcs.push_back(sources[i]);
//     // parameters.Set("srcs", srcs);

//     bool quiet = parameters.Get<bool>("quiet");
//     GraphT graph;
//     // Assign pointers into gunrock graph format
//     // TODO: change to other graph representation, if not using CSR
//     graph.CsrT::Allocate(num_nodes, num_edges, gunrock::util::HOST);
//     graph.CsrT::row_offsets   .SetPointer(row_offsets, num_nodes + 1,
//     gunrock::util::HOST); graph.CsrT::column_indices.SetPointer(col_indices,
//     num_edges, gunrock::util::HOST); graph.CsrT::edge_values
//     .SetPointer(edge_values, num_edges, gunrock::util::HOST);
//     graph.FromCsr(graph.csr(), true, quiet);
//     gunrock::graphio::LoadGraph(parameters, graph);

//     // Run the Template
//     // TODO: add problem specific outputs, e.g.
//     double elapsed_time = gunrock_Template(parameters, graph /*,
//     distances*/);

//     // Cleanup
//     graph.Release();
//     // TODO: problem specific cleanup
//     // srcs.clear();

//     return elapsed_time;
// }

// // Leave this at the end of the file
// // Local Variables:
// // mode:c++
// // c-file-style: "NVIDIA"
// // End:
