// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file knn_app.cu
 *
 * @brief Simple Gunrock Application
 */

// Gunrock api
#include <gunrock/gunrock.h>

// Test utils
#include <gunrock/util/test_utils.cuh>

// Graphio include
#include <gunrock/graphio/graphio.cuh>

// App and test base includes
#include <gunrock/app/app_base.cuh>
#include <gunrock/app/test_base.cuh>

// KNN includes
#include <gunrock/app/knn/knn_enactor.cuh>
#include <gunrock/app/knn/knn_test.cuh>

namespace gunrock {
namespace app {
namespace knn {

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));

  GUARD_CU(parameters.Use<int>(
      "k",
      util::REQUIRED_ARGUMENT | util::MULTI_VALUE | util::OPTIONAL_PARAMETER,
      10, "Numbers of k neighbors.", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "x",
      util::REQUIRED_ARGUMENT | util::MULTI_VALUE | util::OPTIONAL_PARAMETER, 0,
      "Index of reference point.", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "y",
      util::REQUIRED_ARGUMENT | util::MULTI_VALUE | util::OPTIONAL_PARAMETER, 0,
      "Index of reference point.", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<float>(
      "cpu-elapsed", util::REQUIRED_ARGUMENT | util::OPTIONAL_PARAMETER, 0.0f,
      "CPU implementation, elapsed time (ms) for JSON.", __FILE__, __LINE__));

  return retval;
}

/**
 * @brief Run knn tests
 * @tparam     GraphT        Type of the graph
 * @tparam     ValueT        Type of the distances
 * @param[in]  parameters    Excution parameters
 * @param[in]  graph         Input graph
...
 * @param[in]  target        where to perform the app
 * \return hipError_t error message(s), if any
 */
template <typename GraphT>
hipError_t RunTests(util::Parameters &parameters, GraphT &graph,
                     typename GraphT::SizeT k, typename GraphT::SizeT *h_knns,
                     typename GraphT::SizeT *ref_knns, util::Location target) {
  hipError_t retval = hipSuccess;

  typedef typename GraphT::VertexT VertexT;
  typedef typename GraphT::ValueT ValueT;
  typedef typename GraphT::SizeT SizeT;
  typedef Problem<GraphT> ProblemT;
  typedef Enactor<ProblemT> EnactorT;

  // CLI parameters
  bool quiet_mode = parameters.Get<bool>("quiet");
  int num_runs = parameters.Get<int>("num-runs");
  std::string validation = parameters.Get<std::string>("validation");
  util::Info info("knn", parameters, graph);

  VertexT point_x = parameters.Get<int>("x");
  VertexT point_y = parameters.Get<int>("y");

  util::CpuTimer cpu_timer, total_timer;
  cpu_timer.Start();
  total_timer.Start();

  // Allocate problem and enactor on GPU, and initialize them
  ProblemT problem(parameters);
  EnactorT enactor;
  GUARD_CU(problem.Init(graph, k, target));
  GUARD_CU(enactor.Init(problem, target));

  cpu_timer.Stop();
  parameters.Set("preprocess-time", cpu_timer.ElapsedMillis());

  for (int run_num = 0; run_num < num_runs; ++run_num) {
    GUARD_CU(problem.Reset(point_x, point_y, k, target));
    GUARD_CU(enactor.Reset(target));

    util::PrintMsg("__________________________", !quiet_mode);

    cpu_timer.Start();
    GUARD_CU(enactor.Enact());
    cpu_timer.Stop();
    info.CollectSingleRun(cpu_timer.ElapsedMillis());

    util::PrintMsg(
        "--------------------------\nRun " + std::to_string(run_num) +
            " elapsed: " + std::to_string(cpu_timer.ElapsedMillis()) +
            ", #iterations = " +
            std::to_string(enactor.enactor_slices[0].enactor_stats.iteration),
        !quiet_mode);

    if (validation == "each") {
      GUARD_CU(problem.Extract(graph.nodes, k, h_knns));
      SizeT num_errors =
          Validate_Results(parameters, graph, h_knns, ref_knns, false);
    }
  }

  cpu_timer.Start();

  GUARD_CU(problem.Extract(graph.nodes, k, h_knns));
  if (validation == "last") {
    SizeT num_errors =
        Validate_Results(parameters, graph, h_knns, ref_knns, false);
  }

  // compute running statistics
  // Change NULL to problem specific per-vertex visited marker, e.g.
  // h_distances
  info.ComputeTraversalStats(enactor, (VertexT *)NULL);
  // Display_Memory_Usage(problem);
#ifdef ENABLE_PERFORMANCE_PROFILING
  // Display_Performance_Profiling(&enactor);
#endif

  // Clean up
  GUARD_CU(enactor.Release(target));
  GUARD_CU(problem.Release(target));
  cpu_timer.Stop();
  total_timer.Stop();

  info.Finalize(cpu_timer.ElapsedMillis(), total_timer.ElapsedMillis());
  return retval;
}

}  // namespace knn
}  // namespace app
}  // namespace gunrock

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
